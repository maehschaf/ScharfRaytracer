#include "hip/hip_runtime.h"
#pragma once

#include ""
#include "scene.cuh"
#include "hiprand/hiprand_kernel.h"
#include <ctime>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <iostream>

__global__ void initFramebuffer(ViewRender* view) {
	if (blockIdx.x == 0 && threadIdx.x == 0) {
		for (size_t i = 0; i < view->pixelCount; i++) {
			view->frameBuffer[i] = 0.f;
		}
	}
}

__global__ void renderScene(Scene* scene, ViewRender* view) {
	//Get the pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int sample = blockIdx.z * blockDim.z + threadIdx.z;
	if ((x >= view->width) || (y >= view->height) || sample >= view->samples) return;
	int pixelIndex = (y * view->width + x) * CHANNELS;

	//Create a random state
	hiprandState randomState;
	hiprand_init(23984, pixelIndex, 0, &randomState);

	//The sun
	float sunStrength = 14;
	float size = 0.05f;
	float3 sunColor = make_float3(sunStrength);
	float3 sunDir = normalize(make_float3(-1, .5f, .5));
	float sharpness = 2.f;

	float3 finalColor = make_float3(0, 0, 0);
	for (int sample = 0; sample < view->samples; sample++) {

		//What color should be the starting point?
		float3 sampleColor = make_float3(1);
		//Cast our ray
		Ray ray = scene->camera.getRay((x + hiprand_uniform(&randomState) * 2 - 0.5f) / view->width, (y + hiprand_uniform(&randomState) * 2 - 0.5f) / view->height, static_cast<float>(view->height) / view->width);
		Hit hit;
		int bounce = 0;
		for (; bounce < view->maxBounces; bounce++) {
			hit = scene->raytrace(ray, hit.actor);
			if (!hit.hit) {
				//the world background
				float sunAngle = clamp((powf(dot(ray.getDirection(), sunDir), sharpness) - (1 - size)) * (1 / size), 0.f, 1.f);
				sampleColor *= (1 - sunAngle) * scene->background.emissiveColor + sunAngle * sunColor;
				break;
			}
			hit.actor->material.color(hit, ray, sampleColor, &randomState);
			if (hit.actor->material.emissive) break;
		}
		//Failed to hit a light source
		if (bounce >= view->maxBounces) sampleColor *= 0;

	sampleColor /= view->samples;

	//Set the final color
	//finalColor = make_float3((float) x / view->width, (float) y / view->height, 0);
	view->frameBuffer[pixelIndex + 0] += sampleColor.x;
	view->frameBuffer[pixelIndex + 1] += sampleColor.y;
	view->frameBuffer[pixelIndex + 2] += sampleColor.z;
}

int main() {

	std::clock_t startTime = std::clock();

	ViewRender* d_view;
	checkCudaErrors(hipMallocManaged((void**)&d_view, sizeof(ViewRender)));
	//Render settings!
	new (d_view) ViewRender(1920, 1080, 16, 16); //be carefull with placement new....
	d_view->maxBounces = 4;
	d_view->samples = 40;

	Camera camera = Camera({ 0,0,0.3f }, { 1,1,0.2f });

	Sphere lamp = Sphere({ -3, 3, 1 }, Material(make_float3(20, .5f, .5f)), .5f);
	Sphere earth = Sphere({ 0, 0, -2000.f }, Material({ .2f, .9f, .2f }, 1.f), 2000.f);
	Sphere a = Sphere({ 1, 1, 0.2f }, Material({ .8f, .2f, .2f }, 1.f), 0.2f);
	Sphere b = Sphere({ 1, 1.4f, 0.2f }, Material({ .2f, .2f, .8f }, 1.f), 0.2f);
	Sphere c = Sphere({ 5, 4, 1 }, Material({ 1.f, 1.f, 1.f }, 1.f), 1.f);
	Sphere mirror = Sphere({ 2.4f, 2, 0.6f }, Material({ 1.f, 1.f, 1.f }, 0.f), .2f);

	//Background Material
	Material background(make_float3(0.6f, 0.8f, 1.0f) * 0.8f);
	//Material background(make_float3(0));

	Scene* d_scene;
	checkCudaErrors(hipMallocManaged((void**)&d_scene, sizeof(Scene)));
	//Scene setup
	new (d_scene) Scene({ earth, a,b,c, mirror, lamp }, camera, background);

	std::cout << "Samples: " << d_view->samples << " Max Bounces: " << d_view->maxBounces << " Resolution: " << d_view->width << "x" << d_view->height << std::endl;
	std::cout << "Scene Setup time: " << (std::clock() - startTime) / (double)CLOCKS_PER_SEC << "s" << std::endl;
	startTime = std::clock();

	dim3 blocks(d_view->width / d_view->tileSizeX + 1, d_view->height / d_view->tileSizeY + 1, d_view->samples);
	dim3 threads(d_view->tileSizeX, d_view->tileSizeY, 1);

	initFramebuffer << <1, 1 >> > (d_view);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	renderScene << <blocks, threads >> > (d_scene, d_view);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	std::cout << "Render time: " << (std::clock() - startTime) / (double)CLOCKS_PER_SEC << "s" << std::endl;
	startTime = std::clock();

	//Convert float based rgb in frame buffer to only 8bit rgb for the out image
	unsigned char* outPixels = (unsigned char*)malloc(d_view->pixelCount);
	for (size_t i = 0; i < d_view->pixelCount; i++) {
		outPixels[i] = int(clamp(d_view->frameBuffer[i], 0.f, 1.f) * 255);
	}

	stbi_write_jpg("out.jpg", d_view->width, d_view->height, CHANNELS, outPixels, 100);

	std::cout << "Image output time: " << (std::clock() - startTime) / (double)CLOCKS_PER_SEC << "s" << std::endl;

	//Cleanup
	free(outPixels);
	d_view->~ViewRender();
	checkCudaErrors(hipFree(d_view));
	d_scene->~Scene();
	checkCudaErrors(hipFree(d_scene));
}